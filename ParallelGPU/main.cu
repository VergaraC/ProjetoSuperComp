#include "hip/hip_runtime.h"
#include <algorithm>  
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <iomanip>
#include <tuple>


using namespace std;

struct comparar2{
    __host__ __device__
    int operator()(const int& x, const int& y){
        int esq = x - 1;
        int agora = y;

        if (agora > esq && agora > 0){
            return agora;
        }else if(esq>agora && esq>0){
            return esq;
        }else if(agora == esq  && agora > 0){
            return agora;
        }else{
            return 0;
        }
    }
};

struct comparar1{
    int cima;
    int diagonal;
    char c;
    comparar1(int c_) : c(c_){};
    __host__ __device__
    int operator()(const thrust::tuple<char, int, int>& v){

        if(c == thrust::get<0>(v)){
            diagonal = thrust::get<1>(v) + 2;
        }else{
            diagonal = thrust::get<1>(v) - 1;
        }
        cima = thrust::get<2>(v) - 1;
        if ( diagonal > 0 && diagonal > cima ){
            return diagonal;
        }else if( cima > 0 && cima > diagonal){
            return cima;
        }else if(diagonal > 0 && diagonal == cima ){
            return cima; // n importa qual
        }else{
            return 0;
        }

    }

};
int main(){
    int len1, len2;
    string seq1, seq2;
    string seq11, seq22;

    cin >> len1;
    cin >> len2;

    cin >> seq1;
    cin >> seq2;

    seq1 = '0' + seq1;
    seq2 = '0' + seq2;

    int bestScore = -(len1 + len2);
    int scoreAtual;

    thrust::device_vector<char> seq1G(len1+1);
    thrust::device_vector<char> seq2G(len2+1);
    thrust::device_vector<int> mat1(len2+1);
    thrust::device_vector<int> mat2(len2+1);

    //passando pra GPU
    for(int i = 0; i < len1 + 1; i++){
        seq1G[i] = seq1[i];
    }
    for(int i2 = 0; i2 < len2 + 1; i2++){
        seq2G[i2] = seq2[i2];
    }
    //thrust::fill(mat2.begin(), mat2.end(), 0);
    cout << "Preparação GPU feita"<< endl;
    //Prep Substring
    int lenMin = -1;
    int lenMax;
    if(len1 > len2){
        lenMax = len2;
    }else{
        lenMax = len1;
    }
    while(lenMax>= lenMin && lenMax > 1){
            for(int i1 = 0; i1 <= (len1 + 1 - lenMax) ; i1++){
                cout << i1 << endl;
                for(int i2 = 0; i2 <= (len2 + 1 - lenMax); i2++){
                    thrust::fill(mat1.begin(), mat1.end(), 0);

                    for(int i3 = 1; i3 < lenMax+1; i3++){
                        thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(seq2G.begin()+1 + i2, mat1.begin(), mat1.begin()+1)),
                                            thrust::make_zip_iterator(thrust::make_tuple(seq2G.end() + 1 + i2 + lenMax, mat1.begin() + lenMax,mat1.begin() + lenMax + 1)),
                                            mat2.begin() + 1,
                                            comparar1(seq1G[i1]));

                        //cout << "Transform Feito" << endl;

                        thrust::inclusive_scan(mat2.begin(), mat2.begin() + lenMax, mat1.begin(), comparar2());

                        scoreAtual = thrust::reduce(mat1.begin(), mat1.begin() + lenMax, 0, thrust::maximum<int>());
                        
                        if(scoreAtual > bestScore){
                            bestScore = scoreAtual;
                            cout << "New Best: " << bestScore << endl;
                        }
                    }
                }
            }
            lenMax --;
            lenMin = 1 + (bestScore/2); //se n for no min isso ele vais er menor ou igual
            lenMax --;
            cout << "Best Score"<< bestScore << endl;

   
    }
    cout << "Best Score"<< bestScore << endl;
    return 0;
}

